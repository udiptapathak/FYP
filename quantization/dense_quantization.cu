#include "hip/hip_runtime.h"
#include <unordered_map>
#include <up/primitive.h>

static const u16 BUCKET_COUNT = 8096;
const static f32 BUCKET_COUNT_F32 = 8096.l;
const static int DIAMETER = 10;
const static u8 MASK_128 = 1 << 7;


typedef struct {
	f32 lim;
	u8 shouldUpdate;
	f32 base[3];
} Qmap;

static inline __device__ int getBucketIndex(f32 lim, f32 weight) {
	return (int) ((weight - lim) / BUCKET_COUNT_F32) + 1;
}

static inline __device__ f32 getVariance(f32 mean, f32 sum, f32 sumSq,
	const u32 N)
{
	return mean * mean + (2 * mean * sum + sum * sum) / N;
}

std::unordered_map<u8, Qmap> qmaps;

extern "C" {
Qmap QmapNew(f32 inDim)
{
	const f32 lim = 1 / sqrt(inDim); 
	return (Qmap) {lim, 0, {0.l, -lim, lim}};
}



__global__ void QmapApply(Qmap qmap, vp weight_vp, const u32 N)
{
	f32p weight = (f32p) weight_vp;
	u8 cur = 0;
	int node = 0;
	f32p base = qmap.base;
	f32 t_f32;
	const u8 DEPTH = 1;
	for (u32 i = 0 ; i < N ; i++) {
		t_f32 = weight[i];
		while (cur < DEPTH) {
			cur++;
			node = node << 1;
			node += base[node] < t_f32 ? 1 : 2;
		}
		weight[i] = base[node];
	}
}

__global__ void QmapUpdate(Qmap qmap, vp weight_vp, const u32 N)
{
	f32 limLo = - qmap.lim;
	f32 limHi = qmap.lim;
	f32p weight = (f32p) weight_vp;
	u32 bucket[BUCKET_COUNT];
	for (u32 i = 0 ; i < BUCKET_COUNT ; i++)
		bucket[i] = 0;
	for (u32 i = 0 ; i < N ; i++)
		bucket[getBucketIndex(limLo, weight[i])]++;
	u64 sum = 0.l;
	u64 sumSq = 0.l;
	const f32 BUCKET_SIZE= (limHi - limLo) / BUCKET_COUNT;
	f32 t_f32 = limLo - BUCKET_SIZE / 2;
	f32 tb_f32;
	for (u32 i = 0 ; i < BUCKET_COUNT ; i++) {
		t_f32 += BUCKET_SIZE;
		tb_f32 = t_f32 * bucket[i];
		sum += tb_f32;
		sumSq += tb_f32 * tb_f32;
	}
	f32 cur = 0.l;
	f32 curSq = 0.l;
	f32 mean1, mean2, var1, var2;
	f32 minVar = F32_MAX;
	u32 limPoint = 0;
	u32 t1_u32 = 0;
	u32 t2_u32 = 0;
	f32 limMean1, limMean2;
	t_f32 = limLo - BUCKET_SIZE / 2;
	for (u32 i = 0 ; i < BUCKET_COUNT ; i++) {
		t_f32 += BUCKET_SIZE;
		t2_u32 = bucket[i];
		t1_u32 += t2_u32;
		t_f32 = t_f32 * t2_u32;
		cur += tb_f32;
		curSq += tb_f32 * tb_f32;
		mean1 = cur / t1_u32;
		mean2 = (sum - cur) / (N - t1_u32);
		var1 = getVariance(mean1, cur, curSq, t1_u32);
		var2 = getVariance(mean2, sum - cur, sumSq - curSq, N - t1_u32);
		t_f32 = var1 + var2;
		if (t_f32 < minVar) {
			minVar = t_f32;
			limPoint = i;
			limMean1 = mean1;
			limMean2 = mean2;
		}
	}
	qmap.base[0] = weight[limPoint] + F32_MIN;
	qmap.base[1] = limMean1;
	qmap.base[2] = limMean2;
}

bool QmapShouldUpdate(Qmap qmap)
{
	if (qmap.shouldUpdate & MASK_128) {
		qmap.shouldUpdate = 0;
		return true;
	}
	qmap.shouldUpdate++;
	return false;
}

#include <stdio.h>
Qmap QmapLayer(vp layer, u32 inDim)
{
	printf("%p<<\n", layer);
	printf("%u<<\n", (u32) inDim);
	fflush(stdout);
	Qmap ret;
	return ret;
	// auto qmap = qmaps.find(layer);
	// if (qmap != qmaps.end())
	// 	return qmap->second;
	// return qmaps[layer] = QmapNew(inDim);
}

}
